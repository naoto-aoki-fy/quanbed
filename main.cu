#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <stdint.h>

#include <stdexcept>
#include <string>
#include <sstream>
#include <algorithm>
#include <chrono>
#include <random>
#include <utility>
#include <unordered_set>
#include <string_view>
#include <algorithm>

#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cuda/std/complex>
#include <nccl.h>

#include "pipe3.hpp"

#define SQRT2 (1.41421356237309504880168872420969807856967187537694)
#define INV_SQRT2 (1.0/SQRT2)

unsigned int log2_int(unsigned int arg) {
    return sizeof(unsigned int) * CHAR_BIT - __builtin_clz(arg) - 1;
}
unsigned int log2_int(int arg) {
    return log2_int((unsigned int)arg);
}

#if UINT_MAX != ULONG_MAX
// #if sizeof(unsigned int) != sizeof(unsigned long)
unsigned int log2_int(unsigned long arg) {
    return sizeof(unsigned long) * CHAR_BIT - __builtin_clzl(arg) - 1;
}
unsigned int log2_int(long arg) {
    return log2_int((unsigned long)arg);
}
#endif

#if ULONG_MAX != ULLONG_MAX
// #if sizeof(unsigned long) != sizeof(unsigned long long)
unsigned int log2_int(unsigned long long arg) {
    return sizeof(unsigned long long) * CHAR_BIT - __builtin_clzll(arg) - 1;
}
unsigned int log2_int(long long arg) {
    return log2_int((unsigned long long)arg);
}
#endif

typedef double my_float_t;
typedef cuda::std::complex<my_float_t> my_complex_t;

constexpr const char* get_filename(const char* filename_abs) {
    size_t const pos = std::string_view(filename_abs).rfind("/");
    return (pos != std::string_view::npos) ? &filename_abs[pos+1] : filename_abs;
}

template <typename Func>
void check_cuda(char const* const filename, int const lineno, char const* const funcname, Func func)
{
    auto err = func();
    if (err != hipSuccess)
    {
        fprintf(stderr, "[debug] %s:%d call:%s error:%s\n", filename, lineno, funcname, hipGetErrorString(err));
        exit(1);
    }
}

#define CHECK_CUDA(func, ...) check_cuda(get_filename(__FILE__), __LINE__, #func "(" #__VA_ARGS__ ")", [&](){return func(__VA_ARGS__);})

#define CASE_RETURN(code) case code: return #code

static const char *curandGetErrorString(hiprandStatus_t error) {
    switch (error) {
        CASE_RETURN(HIPRAND_STATUS_SUCCESS);
        CASE_RETURN(HIPRAND_STATUS_VERSION_MISMATCH);
        CASE_RETURN(HIPRAND_STATUS_NOT_INITIALIZED);
        CASE_RETURN(HIPRAND_STATUS_ALLOCATION_FAILED);
        CASE_RETURN(HIPRAND_STATUS_TYPE_ERROR);
        CASE_RETURN(HIPRAND_STATUS_OUT_OF_RANGE);
        CASE_RETURN(HIPRAND_STATUS_LENGTH_NOT_MULTIPLE);
        CASE_RETURN(HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED);
        CASE_RETURN(HIPRAND_STATUS_LAUNCH_FAILURE);
        CASE_RETURN(HIPRAND_STATUS_PREEXISTING_FAILURE);
        CASE_RETURN(HIPRAND_STATUS_INITIALIZATION_FAILED);
        CASE_RETURN(HIPRAND_STATUS_ARCH_MISMATCH);
        CASE_RETURN(HIPRAND_STATUS_INTERNAL_ERROR);
    }
    return "<unknown>";
}

template <typename Func>
void check_curand(char const* const filename, int const lineno, char const* const funcname, Func func)
{
    auto err = func();
    if (err != HIPRAND_STATUS_SUCCESS)
    {
        fprintf(stderr, "[debug] %s:%d call:%s error:%s\n", filename, lineno, funcname, curandGetErrorString(err));
        exit(1);
    }
}

#define CHECK_CURAND(func, ...) check_curand(get_filename(__FILE__), __LINE__, #func "(" #__VA_ARGS__ ")", [&](){return func(__VA_ARGS__);})

template <typename Func>
void check_nccl(char const* const filename, int const lineno, char const* const funcname, Func func)
{
    auto err = func();
    if (err != ncclSuccess)
    {
        fprintf(stderr, "[debug] %s:%d call:%s error:%s\n", filename, lineno, funcname, ncclGetErrorString(err));
        exit(1);
    }
}

#define CHECK_NCCL(func, ...) check_nccl(get_filename(__FILE__), __LINE__, #func "(" #__VA_ARGS__ ")", [&](){return func(__VA_ARGS__);})

template <typename Func>
class Defer {
public:
    Defer(Func func) : func_(func) {}
    ~Defer() { this->func_(); }
private:
    Func func_;
};

#define CONCAT(a, b) CONCAT_INNER(a, b)
#define CONCAT_INNER(a, b) a ## b
#define UNIQUE_NAME(base) CONCAT(base, __LINE__)

#define DEFER_CHECK_CUDA(func, ...) Defer UNIQUE_NAME(defer_)([&](){ CHECK_CUDA(func, __VA_ARGS__);})

#define DEFER_CODE(code) Defer UNIQUE_NAME(defer_)([&]()code)

__global__ void norm_sum_reduce_kernel(my_complex_t const* const input_global, my_float_t* const output_global)
{
    extern __shared__ my_float_t sum_shared[];
    int64_t const idx =  blockDim.x * blockIdx.x + threadIdx.x;
    sum_shared[threadIdx.x] = cuda::std::norm(input_global[idx]);

    my_float_t sum_cached;
    sum_cached = sum_shared[threadIdx.x];
    for(int active_threads = blockDim.x; active_threads > 1;) {
        int const half_active_threads = active_threads >> 1;
        active_threads = (active_threads + 1) >> 1;
        __syncthreads();
        if(threadIdx.x < half_active_threads){
            sum_cached += sum_shared[threadIdx.x + active_threads];
            sum_shared[threadIdx.x] = sum_cached;
        }
    }
    if (threadIdx.x == 0) {
        output_global[blockIdx.x] = sum_shared[0];
    }
}

__global__ void sum_reduce_kernel(my_float_t const* const input_global, my_float_t* const output_global)
{
    extern __shared__ my_float_t sum_shared[];
    int64_t const idx =  blockDim.x * blockIdx.x + threadIdx.x;
    sum_shared[threadIdx.x] = input_global[idx];

    my_float_t sum_cached;
    sum_cached = sum_shared[threadIdx.x];
    for(int active_threads = blockDim.x; active_threads > 1;) {
        int const half_active_threads = active_threads >> 1;
        active_threads = (active_threads + 1) >> 1;
        __syncthreads();
        if(threadIdx.x < half_active_threads){
            sum_cached += sum_shared[threadIdx.x + active_threads];
            sum_shared[threadIdx.x] = sum_cached;
        }
    }
    if (threadIdx.x == 0) {
        output_global[blockIdx.x] = sum_shared[0];
    }
}

__global__ void normalize_kernel(my_float_t* const data_global, my_float_t const factor)
{
    int64_t const idx = blockDim.x * blockIdx.x + threadIdx.x;
    data_global[idx] *= factor;
}

class hadamard { public:
    static __device__ __host__ void apply(int64_t const thread_num, int64_t const num_qubits, int64_t const target_qubit_num, my_complex_t* const state_data_device) {

        uint64_t const lower_mask = (((uint64_t)1)<<target_qubit_num) - (uint64_t)1;

        int64_t const index_state_lower = thread_num & lower_mask;
        int64_t const index_state_higher = (thread_num & ~lower_mask) << ((int64_t)1);

        int64_t const index_state_0 = index_state_lower | index_state_higher;
        int64_t const index_state_1 = index_state_0 | (((int64_t)1)<<target_qubit_num);

        my_complex_t const amp_state_0 = state_data_device[index_state_0];
        my_complex_t const amp_state_1 = state_data_device[index_state_1];

        state_data_device[index_state_0] = (amp_state_0 + amp_state_1) * INV_SQRT2;
        state_data_device[index_state_1] = (amp_state_0 - amp_state_1) * INV_SQRT2;

    }
};

template<class Gate>
__global__ void cuda_gate(int64_t const num_qubits, int64_t const target_qubit_num, my_complex_t* const state_data_device) {
    int64_t const thread_num = threadIdx.x + blockIdx.x * blockDim.x;
    Gate::apply(thread_num, num_qubits, target_qubit_num, state_data_device);
}


int main(int argc, char** argv) {

    float elapsed_ms, elapsed_ms_2;

    setvbuf(stdout, NULL, _IOLBF, 1024 * 512);

    MPI_Init(&argc, &argv);

    int num_procs, proc_num;

    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);
    MPI_Comm_rank(MPI_COMM_WORLD, &proc_num);

    if (proc_num==0) {
        fprintf(stderr, "[info] num_procs=%d\n", num_procs);
    }

    int const gpu_id = proc_num;
    // int const gpu_id = 0;
    CHECK_CUDA(hipSetDevice, gpu_id);

    ncclUniqueId nccl_id;
    if (proc_num == 0) {
        CHECK_NCCL(ncclGetUniqueId, &nccl_id);
    }

    ncclComm_t nccl_comm;
    MPI_Bcast(&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD);
    int nccl_rank = proc_num;
    CHECK_NCCL(ncclCommInitRank, &nccl_comm, num_procs, nccl_id, nccl_rank);

    int const num_qubits = 24;
    if (proc_num == 0) { fprintf(stderr, "[info] num_qubits=%d\n", num_qubits); }



    std::vector<int> perm_p2l(num_qubits);
    std::vector<int> perm_l2p(num_qubits);

    for(int qubit_num=0; qubit_num<num_qubits; qubit_num++) {
        perm_p2l[qubit_num] = qubit_num;
        perm_l2p[qubit_num] = qubit_num;
    }

    int const num_samples = 32;
    int const rng_seed = 12345;

    int const log_num_procs = log2_int(num_procs);

    int const log_block_size = 8;
    int const target_qubit_num_begin = 0;
    // int const target_qubit_num_end = 0;
    int const target_qubit_num_end = num_qubits;

    if (proc_num == 0) { fprintf(stderr, "[info] log_block_size=%d\n", log_block_size); }

    hipStream_t stream;
    hipEvent_t event_1;
    hipEvent_t event_2;

    CHECK_CUDA(hipStreamCreate, &stream);
    DEFER_CHECK_CUDA(hipStreamDestroy, stream);

    CHECK_CUDA(hipEventCreateWithFlags, &event_1, hipEventDefault);
    DEFER_CHECK_CUDA(hipEventDestroy, event_1);

    CHECK_CUDA(hipEventCreateWithFlags, &event_2, hipEventDefault);
    DEFER_CHECK_CUDA(hipEventDestroy, event_2);

    int64_t const num_states = ((int64_t)1) << ((int64_t)num_qubits);

    int const num_qubits_local = num_qubits - log_num_procs;
    
    int64_t const num_states_local = ((int64_t)1) << ((int64_t)num_qubits_local);
    int const block_size = 1 << log_block_size;
    int64_t const num_blocks = ((int64_t)1) << ((int64_t)(num_qubits_local - 1 - log_block_size));

    if (proc_num == 0) { fprintf(stderr, "[info] malloc device memory\n"); }

    my_complex_t* state_data_device;
    CHECK_CUDA(hipMalloc, &state_data_device, num_states_local * sizeof(*state_data_device));
    DEFER_CHECK_CUDA(hipFree, state_data_device);

    int const log_swap_buffer_total_length = (num_qubits_local>30)? num_qubits_local - 3 : num_qubits_local;
    uint64_t const swap_buffer_total_length = UINT64_C(1) << log_swap_buffer_total_length;
    my_complex_t* swap_buffer;
    CHECK_CUDA(hipMalloc, &swap_buffer, swap_buffer_total_length * sizeof(my_complex_t));
    DEFER_CHECK_CUDA(hipFree, swap_buffer);

    my_float_t* norm_sum_device;
    CHECK_CUDA(hipMalloc, &norm_sum_device, (num_states_local>>log_block_size) * sizeof(my_float_t));
    // DEFER_CHECK_CUDA(hipFree, norm_sum_device);

    if (proc_num == 0) { fprintf(stderr, "[info] generating random state\n"); }
    hiprandGenerator_t rng_device;

    CHECK_CURAND(hiprandCreateGenerator, &rng_device, HIPRAND_RNG_PSEUDO_DEFAULT);
    CHECK_CURAND(hiprandSetStream, rng_device, stream);
    // CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device, rng_seed + proc_num);

    if (proc_num == 0) { fprintf(stderr, "[info] gpu reduce\n"); } 
    CHECK_CUDA(hipEventRecord, event_1, stream);

    CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device, rng_seed + proc_num);
    CHECK_CURAND(hiprandGenerateNormalDouble, rng_device, (my_float_t*)(void*)state_data_device, num_states_local * 2, 0.0, 1.0);

    // CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device, rng_seed + proc_num * 2);
    // CHECK_CURAND(hiprandGenerateNormalDouble, rng_device, (my_float_t*)(void*)state_data_device, num_states_local, 0.0, 1.0);

    // hiprandGenerator_t rng_device_2;

    // CHECK_CURAND(hiprandCreateGenerator, &rng_device_2, HIPRAND_RNG_PSEUDO_DEFAULT);
    // CHECK_CURAND(hiprandSetStream, rng_device_2, stream);
    // CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device_2, rng_seed + proc_num * 2 + 1);

    // CHECK_CURAND(hiprandGenerateNormalDouble, rng_device_2, &((my_float_t*)(void*)state_data_device)[num_states_local], num_states_local, 0.0, 1.0);

    {
        int64_t data_length = num_states_local;
        int64_t num_blocks_reduce;
        int block_size_reduce;

        if (data_length > block_size) {
            block_size_reduce = block_size;
            num_blocks_reduce = data_length >> log_block_size;
        } else {
            block_size_reduce = data_length;
            num_blocks_reduce = 1;
        }

        norm_sum_reduce_kernel<<<num_blocks_reduce, block_size_reduce, sizeof(my_float_t) * block_size_reduce, stream>>>(state_data_device, norm_sum_device);

        data_length = num_blocks_reduce;

        while (data_length > 1) {
            if (data_length > block_size) {
                block_size_reduce = block_size;
                num_blocks_reduce = data_length >> log_block_size;
            } else {
                block_size_reduce = data_length;
                num_blocks_reduce = 1;
            }

            sum_reduce_kernel<<<num_blocks_reduce, block_size_reduce, sizeof(my_float_t) * block_size_reduce, stream>>>(norm_sum_device, norm_sum_device);

            data_length = num_blocks_reduce;
        }
    }

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    my_float_t norm_sum_local;
    CHECK_CUDA(hipMemcpyAsync, &norm_sum_local, norm_sum_device, sizeof(my_float_t), hipMemcpyDeviceToHost, stream);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    CHECK_CUDA(hipFree, (void*)norm_sum_device);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    CHECK_CUDA(hipStreamSynchronize, stream);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    my_float_t norm_sum_global;
    MPI_Allreduce(&norm_sum_local, &norm_sum_global, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    if (proc_num == 0) { fprintf(stderr, "[info] norm_sum_global=%lf\n", norm_sum_global); }

    if (proc_num == 0) { fprintf(stderr, "[info] normalize\n"); }

    my_float_t const normalize_factor = 1.0 / sqrt(norm_sum_global);
    // fprintf(stderr, "[debug] normalize_factor=%.20e\n", normalize_factor);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    normalize_kernel<<<1ULL<<(num_qubits_local+1-log_block_size), block_size, 0, stream>>>((my_float_t*)(void*)state_data_device, normalize_factor);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    CHECK_CUDA(hipEventRecord, event_2, stream);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    CHECK_CUDA(hipStreamSynchronize, stream);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    CHECK_CUDA(hipEventElapsedTime, &elapsed_ms, event_1, event_2);

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    MPI_Reduce(&elapsed_ms, &elapsed_ms_2, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
    elapsed_ms = elapsed_ms_2;

    // fprintf(stderr, "[debug] line=%d\n", __LINE__);

    if(proc_num==0) {
        fprintf(stderr, "[info] rng elapsed=%lf\n", elapsed_ms * 1e-3);
        fprintf(stderr, "[info] normalize done\n");
        fprintf(stderr, "[info] gpu_hadamard\n");
    }

    MPI_Barrier(MPI_COMM_WORLD);

    for(int sample_num=0; sample_num < num_samples; ++sample_num) {

        CHECK_CUDA(hipEventRecord, event_1, stream);

        for(int target_qubit_num_logical = target_qubit_num_begin; target_qubit_num_logical < target_qubit_num_end; target_qubit_num_logical++) {

            int target_qubit_num_physical = perm_l2p[target_qubit_num_logical];
            // if(proc_num==0) fprintf(stderr, "[debug] target_qubit_num_logical=%d target_qubit_num_physical=%d\n", target_qubit_num_logical, target_qubit_num_physical);
            // MPI_Barrier(MPI_COMM_WORLD);

            /* target qubits is global */
            if (target_qubit_num_physical >= num_qubits_local) {

                int const* const swap_target_global_list = &target_qubit_num_physical;
                int const swap_target_local = num_qubits - log_num_procs - 1;
                int const* const swap_target_local_list = &swap_target_local;
                int const num_targets = 1;

                // b_min
                int const swap_target_local_min = *std::min_element(swap_target_local_list, swap_target_local_list + num_targets);

                uint64_t const local_buf_length = UINT64_C(1) << swap_target_local_min;
                uint64_t swap_buffer_length = swap_buffer_total_length;
                if (swap_buffer_length > local_buf_length) {
                    swap_buffer_length = local_buf_length;
                }

                // generate a mask for generating global_nonswap_self
                uint64_t global_swap_self_mask = 0;
                for (int target_num = 0; target_num < num_targets; target_num++) {
                    // a_delta = a – n_local
                    int const swap_target_global_delta = swap_target_global_list[target_num] - num_qubits_local;
                    global_swap_self_mask |= (UINT64_C(1) << swap_target_global_delta);
                }

                // global_nonswap_self = make proc_num_self's a_delta_i-th digit zero
                uint64_t const global_nonswap_self = proc_num & ~global_swap_self_mask;

                // 1<<(num_local_qubits - b_min) 
                uint64_t const num_local_areas = UINT64_C(1) << (num_qubits_local - swap_target_local_min);
                for (uint64_t local_num_self = 0; local_num_self < num_local_areas; local_num_self++) {

                    // global_swap_peer = OR_i (local_num_selfのb_delta_i桁目)をa_delta_i桁目にする
                    uint64_t global_swap_peer = 0;
                    for (int target_num = 0; target_num < num_targets; target_num++) {
                        // a_delta_i
                        int const swap_target_global_delta = swap_target_global_list[target_num] - num_qubits_local;
                        // b_delta_i
                        int const swap_target_local_delta = swap_target_local_list[target_num] - swap_target_local_min;
                        global_swap_peer |=
                            // local_num_selfのb_delta_i桁目
                            ((local_num_self >> swap_target_local_delta) & 1)
                            // をa_delta_i桁目にする
                            << swap_target_global_delta;
                        
                    }

                    uint64_t const proc_num_peer = global_swap_peer | global_nonswap_self;

                    // send & recv
                    if (proc_num_peer == proc_num) { continue; }
                    // CHECK_NCCL(ncclSend, &state_data_device[local_num_self * local_buf_length], local_buf_length, ncclDouble, proc_num_peer, nccl_comm, stream);
                    // CHECK_NCCL(ncclRecv, &state_data_device[local_num_self * local_buf_length], local_buf_length, ncclDouble, proc_num_peer, nccl_comm, stream);
                    bool is_peer_greater = proc_num_peer > proc_num;
                    for (uint64_t buffer_pos = 0; buffer_pos < local_buf_length; buffer_pos += swap_buffer_length) {
                        CHECK_NCCL(ncclGroupStart);
                        for (int send_recv = 0; send_recv < 2; send_recv++) {
                            if (send_recv ^ is_peer_greater) {
                                CHECK_NCCL(ncclSend, &state_data_device[local_num_self * local_buf_length + buffer_pos], swap_buffer_length * 2 /* complex */, ncclDouble, proc_num_peer, nccl_comm, stream);
                            } else {
                                CHECK_NCCL(ncclRecv, swap_buffer, swap_buffer_length * 2 /* complex */, ncclDouble, proc_num_peer, nccl_comm, stream);
                            }
                        }
                        CHECK_NCCL(ncclGroupEnd);
                        CHECK_CUDA(hipMemcpyAsync, &state_data_device[local_num_self * local_buf_length + buffer_pos], swap_buffer, swap_buffer_length * sizeof(my_complex_t), hipMemcpyDeviceToDevice, stream);
                    }

                }

                // swap_target_global_logical_list[:] = perm_p2l[swap_target_global_list[:]]
                // swap_target_local_logical_list[:] = perm_p2l[swap_target_local_list[:]]
                std::vector<int> swap_target_local_logical_list(num_targets);
                std::vector<int> swap_target_global_logical_list(num_targets);
                for (int target_num = 0; target_num < num_targets; target_num++) {
                    swap_target_local_logical_list[target_num] = perm_p2l[swap_target_local_list[target_num]];
                    swap_target_global_logical_list[target_num] = perm_p2l[swap_target_global_list[target_num]];
                }

                // update p2l & l2p
                // perm_p2l[swap_target_global_list[:]] = swap_target_local_logical_list[:]
                // perm_p2l[swap_target_local_list[:]] = swap_target_global_logical_list[:]
                // perm_l2p[swap_target_global_logical_list[:]] = swap_target_local_list[:]
                // perm_l2p[swap_target_local_logical_list[:]] = swap_target_global_list[:]

                for (int target_num = 0; target_num < num_targets; target_num++) {
                    perm_p2l[swap_target_global_list[target_num]] = swap_target_local_logical_list[target_num];
                    perm_p2l[swap_target_local_list[target_num]] = swap_target_global_logical_list[target_num];
                    perm_l2p[swap_target_global_logical_list[target_num]] = swap_target_local_list[target_num];
                    perm_l2p[swap_target_local_logical_list[target_num]] = swap_target_global_list[target_num];
                }

                target_qubit_num_physical = swap_target_local;

            }

            cuda_gate<hadamard><<<num_blocks, block_size, 0, stream>>>(num_qubits, target_qubit_num_physical, state_data_device);
        }

        CHECK_CUDA(hipEventRecord, event_2, stream);

        CHECK_CUDA(hipStreamSynchronize, stream);

        CHECK_CUDA(hipEventElapsedTime, &elapsed_ms, event_1, event_2);
        MPI_Reduce(&elapsed_ms, &elapsed_ms_2, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
        elapsed_ms = elapsed_ms_2;
        if (proc_num == 0) {
            fprintf(stderr, "[info] elapsed_gpu=%f\n", elapsed_ms * 1e-3);
            fprintf(stdout, "%lf\n", elapsed_ms * 1e-3);
        }

    }

    if(false) {
        if(proc_num==0) {
            fprintf(stderr, "[info] gathering state data\n");

            process cksumproc;
            char const* const cksumproc_argv[] = {"openssl", "sha256", "-r", NULL};
            if (popen3(&cksumproc, cksumproc_argv, true, true, false) != 0) {
                fprintf(stderr, "[errpr] popen3 failed\n");
                exit(1);
            }

            my_complex_t* state_data_host = (my_complex_t*)malloc(num_states * sizeof(my_complex_t));
            DEFER_CODE({free(state_data_host);});

            CHECK_CUDA(hipMemcpyAsync, state_data_host, state_data_device, num_states_local * sizeof(my_complex_t), hipMemcpyDeviceToHost, stream);
            for(int peer_proc_num=1; peer_proc_num<num_procs; peer_proc_num++) {
                MPI_Status mpi_status;
                MPI_Recv(&state_data_host[peer_proc_num * num_states_local], num_states_local * 2, MPI_DOUBLE, peer_proc_num, 0, MPI_COMM_WORLD, &mpi_status);
            }
            CHECK_CUDA(hipStreamSynchronize, stream);

            for(int64_t state_num_logical = 0; state_num_logical < num_states; state_num_logical++) {
                int64_t state_num_physical = 0;
                for(int qubit_num_logical = 0; qubit_num_logical < num_qubits; qubit_num_logical++) {
                    int qubit_num_physical = perm_l2p[qubit_num_logical];
                    state_num_physical = state_num_physical | (((state_num_logical >> qubit_num_logical) & 1) << qubit_num_physical);
                }
                fwrite(&state_data_host[state_num_physical], sizeof(my_complex_t), 1, cksumproc.stdin);
            }
            // for(int64_t state_num_logical = 0; state_num_logical < num_states; state_num_logical++) {
            //     int64_t state_num_physical = state_num_logical;
            //     fwrite(&state_data_host[state_num_physical], sizeof(my_complex_t), 1, cksumproc.stdin);
            // }
            // fprintf(stderr, "[debug] line=%d\n", __LINE__);

            fclose(cksumproc.stdin);

            int cksumbuf_length = 128;
            char cksumbuf[cksumbuf_length];
            fread(cksumbuf, 1, cksumbuf_length, cksumproc.stdout);
            char* cksum_space_pos = strchr(cksumbuf, ' ');
            if (cksum_space_pos!=NULL) {
                *cksum_space_pos = '\0';
            }
            fprintf(stderr, "[info] check sum: %s\n", cksumbuf);

            // fprintf(stderr, "[debug] line=%d\n", __LINE__);

            int cksumproc_status;
            waitpid(cksumproc.pid, &cksumproc_status, 0);

            // fprintf(stderr, "[debug] line=%d\n", __LINE__);

            if (cksumproc_status!=0) {
                fprintf(stderr, "[warn] cksumproc_status=%d\n", cksumproc_status);
            }

            // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        } else {
            // for(int proc_num = 1; proc_num < num_procs; proc_num++) {
            MPI_Send(state_data_device, num_states_local * 2, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
            // }
        }
    }

    MPI_Finalize();

    return 0;

}