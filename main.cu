#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <stdint.h>

#include <stdexcept>
#include <string>
#include <sstream>
#include <algorithm>
#include <chrono>
#include <random>
#include <utility>
#include <unordered_set>
#include <string_view>
#include <algorithm>
#include <string>
#include <unordered_set>
#include <tuple>

#include <openssl/evp.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <cuda/std/complex>
#include <nccl.h>

#include "log2_int.hpp"
#include "check_x.hpp"
#include "group_by_hostname.hpp"
#include "reorder_macro.h"

#define SQRT2 (1.41421356237309504880168872420969807856967187537694)
#define INV_SQRT2 (1.0/SQRT2)

typedef double my_float_t;
typedef cuda::std::complex<my_float_t> my_complex_t;

__global__ void norm_sum_reduce_kernel(my_complex_t const* const input_global, my_float_t* const output_global)
{
    extern __shared__ my_float_t sum_shared[];
    int64_t const idx =  blockDim.x * blockIdx.x + threadIdx.x;
    sum_shared[threadIdx.x] = cuda::std::norm(input_global[idx]);

    my_float_t sum_cached;
    sum_cached = sum_shared[threadIdx.x];
    for(int active_threads = blockDim.x; active_threads > 1;) {
        int const half_active_threads = active_threads >> 1;
        active_threads = (active_threads + 1) >> 1;
        __syncthreads();
        if(threadIdx.x < half_active_threads){
            sum_cached += sum_shared[threadIdx.x + active_threads];
            sum_shared[threadIdx.x] = sum_cached;
        }
    }
    if (threadIdx.x == 0) {
        output_global[blockIdx.x] = sum_shared[0];
    }
}

__global__ void sum_reduce_kernel(my_float_t const* const input_global, my_float_t* const output_global)
{
    extern __shared__ my_float_t sum_shared[];
    int64_t const idx =  blockDim.x * blockIdx.x + threadIdx.x;
    sum_shared[threadIdx.x] = input_global[idx];

    my_float_t sum_cached;
    sum_cached = sum_shared[threadIdx.x];
    for(int active_threads = blockDim.x; active_threads > 1;) {
        int const half_active_threads = active_threads >> 1;
        active_threads = (active_threads + 1) >> 1;
        __syncthreads();
        if(threadIdx.x < half_active_threads){
            sum_cached += sum_shared[threadIdx.x + active_threads];
            sum_shared[threadIdx.x] = sum_cached;
        }
    }
    if (threadIdx.x == 0) {
        output_global[blockIdx.x] = sum_shared[0];
    }
}

__global__ void normalize_kernel(my_float_t* const data_global, my_float_t const factor)
{
    int64_t const idx = blockDim.x * blockIdx.x + threadIdx.x;
    data_global[idx] *= factor;
}

class hadamard { public:
    static __device__ __host__ void apply(int64_t const thread_num, int64_t const num_qubits, int64_t const target_qubit_num, my_complex_t* const state_data_device) {

        uint64_t const lower_mask = (((uint64_t)1)<<target_qubit_num) - (uint64_t)1;

        int64_t const index_state_lower = thread_num & lower_mask;
        int64_t const index_state_higher = (thread_num & ~lower_mask) << ((int64_t)1);

        int64_t const index_state_0 = index_state_lower | index_state_higher;
        int64_t const index_state_1 = index_state_0 | (((int64_t)1)<<target_qubit_num);

        my_complex_t const amp_state_0 = state_data_device[index_state_0];
        my_complex_t const amp_state_1 = state_data_device[index_state_1];

        state_data_device[index_state_0] = (amp_state_0 + amp_state_1) * INV_SQRT2;
        state_data_device[index_state_1] = (amp_state_0 - amp_state_1) * INV_SQRT2;

    }
};

template<class Gate>
__global__ void cuda_gate(int64_t const num_qubits, int64_t const target_qubit_num, my_complex_t* const state_data_device) {
    int64_t const thread_num = threadIdx.x + blockIdx.x * blockDim.x;
    Gate::apply(thread_num, num_qubits, target_qubit_num, state_data_device);
}

int main(int argc, char** argv) {

    // **注意**: normalize_factorが並列方法によって若干計算結果に違いがあるので、ノーマライズしてしまうと、チェックサムが一致しなくなる
    // **Note**: The `normalize_factor` may cause slight differences in calculation results due to parallel processing methods. As a result, normalization can lead to a mismatch in the checksum.
    bool const do_normalization = false;
    bool const calc_checksum = false;
    int const num_rand_areas = 1;
    bool const use_unified_memory = false;

    float elapsed_ms, elapsed_ms_2;

    setvbuf(stdout, NULL, _IOLBF, 1024 * 512);

    MPI_Init(&argc, &argv);

    int num_procs, proc_num;

    MPI_Comm_size(MPI_COMM_WORLD, &num_procs);
    MPI_Comm_rank(MPI_COMM_WORLD, &proc_num);

    if (proc_num==0) {
        fprintf(stderr, "[info] num_procs=%d\n", num_procs);
    }

    /* ==== begin local rank ==== */
    auto [my_hostname, my_node_number, my_node_local_rank, node_count] = group_by_host(proc_num, num_procs);
    fprintf(stderr,
            "[debug] Rank %d on host %s -> assigned node number: %d, local node rank: %d (total nodes: %d)\n",
            proc_num, my_hostname.c_str(), my_node_number, my_node_local_rank, node_count);
    // MPI_Finalize();
    // return 0;
    /* ==== end local rank ==== */

    // int const gpu_id = proc_num;
    int const gpu_id = my_node_local_rank;
    // int const gpu_id = 0;
    CHECK_CUDA(hipSetDevice, gpu_id);

    ncclUniqueId nccl_id;
    if (proc_num == 0) {
        CHECK_NCCL(ncclGetUniqueId, &nccl_id);
    }

    ncclComm_t nccl_comm;
    MPI_Bcast(&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD);
    int nccl_rank = proc_num;
    CHECK_NCCL(ncclCommInitRank, &nccl_comm, num_procs, nccl_id, nccl_rank);

    int const num_qubits = 24;
    if (proc_num == 0) { fprintf(stderr, "[info] num_qubits=%d\n", num_qubits); }

    std::vector<int> perm_p2l(num_qubits);
    std::vector<int> perm_l2p(num_qubits);

    for(int qubit_num=0; qubit_num<num_qubits; qubit_num++) {
        perm_p2l[qubit_num] = qubit_num;
        perm_l2p[qubit_num] = qubit_num;
    }

    int const num_samples = 64;
    int const rng_seed = 12345;

    int const log_num_procs = log2_int(num_procs);

    int const log_block_size = 8;
    int const target_qubit_num_begin = 0;
    // int const target_qubit_num_end = 0;
    int const target_qubit_num_end = num_qubits;

    if (proc_num == 0) { fprintf(stderr, "[info] log_block_size=%d\n", log_block_size); }

    hipStream_t stream;
    hipEvent_t event_1;
    hipEvent_t event_2;

    CHECK_CUDA(hipStreamCreate, &stream);
    DEFER_CHECK_CUDA(hipStreamDestroy, stream);

    CHECK_CUDA(hipEventCreateWithFlags, &event_1, hipEventDefault);
    DEFER_CHECK_CUDA(hipEventDestroy, event_1);

    CHECK_CUDA(hipEventCreateWithFlags, &event_2, hipEventDefault);
    DEFER_CHECK_CUDA(hipEventDestroy, event_2);

    int64_t const num_states = ((int64_t)1) << ((int64_t)num_qubits);

    int const num_qubits_local = num_qubits - log_num_procs;
    
    int64_t const num_states_local = ((int64_t)1) << ((int64_t)num_qubits_local);
    int const block_size = 1 << log_block_size;
    int64_t const num_blocks = ((int64_t)1) << ((int64_t)(num_qubits_local - 1 - log_block_size));

    if (proc_num == 0) { fprintf(stderr, "[info] malloc device memory\n"); }

    my_complex_t* state_data_device;
    if (use_unified_memory) {
        CHECK_CUDA(hipMallocManaged, &state_data_device, num_states_local * sizeof(*state_data_device));
        CHECK_CUDA(hipMemAdvise, state_data_device, num_states_local * sizeof(*state_data_device), hipMemAdviseSetPreferredLocation, gpu_id);
    } else {
        CHECK_CUDA(hipMalloc, &state_data_device, num_states_local * sizeof(*state_data_device));
    }
    DEFER_CHECK_CUDA(hipFree, state_data_device);

    int const log_swap_buffer_total_length = (num_qubits_local>30)? num_qubits_local - 3 : num_qubits_local;
    // int const log_swap_buffer_total_length = num_qubits_local;
    uint64_t const swap_buffer_total_length = UINT64_C(1) << log_swap_buffer_total_length;
    my_complex_t* swap_buffer;
    CHECK_CUDA(hipMalloc, &swap_buffer, swap_buffer_total_length * sizeof(my_complex_t));
    // CHECK_CUDA(hipMallocManaged, &swap_buffer, swap_buffer_total_length * sizeof(my_complex_t));
    DEFER_CHECK_CUDA(hipFree, swap_buffer);

    my_float_t* norm_sum_device;
    CHECK_CUDA(hipMalloc, &norm_sum_device, (num_states_local>>log_block_size) * sizeof(my_float_t));
    // DEFER_CHECK_CUDA(hipFree, norm_sum_device);

    if (proc_num == 0) { fprintf(stderr, "[info] generating random state\n"); }
    hiprandGenerator_t rng_device;

    // CHECK_CURAND(hiprandCreateGenerator, &rng_device, HIPRAND_RNG_PSEUDO_DEFAULT);
    // CHECK_CURAND(hiprandSetStream, rng_device, stream);
    // CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device, rng_seed + proc_num);

    CHECK_CUDA(hipEventRecord, event_1, stream);

    // if (false) {
    //     CHECK_CURAND(hiprandCreateGenerator, &rng_device, HIPRAND_RNG_PSEUDO_DEFAULT);
    //     CHECK_CURAND(hiprandSetStream, rng_device, stream);
    //     CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device, rng_seed + proc_num);
    //     CHECK_CURAND(hiprandGenerateNormalDouble, rng_device, (my_float_t*)(void*)state_data_device, num_states_local * 2 /* complex */, 0.0, 1.0);
    //     CHECK_CURAND(hiprandDestroyGenerator, rng_device);
    // } else
    {
        // int const num_rand_areas = 4;
        int const log_num_rand_areas = log2_int(num_rand_areas);
        // if (log_num_rand_areas!=1) { throw; }
        uint64_t const num_states_rand_area = num_states_local >> log_num_rand_areas;
        for (int rand_area_num = 0; rand_area_num < num_rand_areas; rand_area_num++) {
            CHECK_CURAND(hiprandCreateGenerator, &rng_device, HIPRAND_RNG_PSEUDO_DEFAULT);
            CHECK_CURAND(hiprandSetStream, rng_device, stream);
            CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device, rng_seed + proc_num * num_rand_areas + rand_area_num);
            CHECK_CURAND(hiprandGenerateNormalDouble, rng_device, (my_float_t*)(void*)(state_data_device + num_states_rand_area * ((uint64_t)rand_area_num)), num_states_rand_area * 2 /* complex */, 0.0, 1.0);
            CHECK_CURAND(hiprandDestroyGenerator, rng_device);
        }
    }
    // CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device, rng_seed + proc_num * 2);
    // CHECK_CURAND(hiprandGenerateNormalDouble, rng_device, (my_float_t*)(void*)state_data_device, num_states_local, 0.0, 1.0);

    // hiprandGenerator_t rng_device_2;

    // CHECK_CURAND(hiprandCreateGenerator, &rng_device_2, HIPRAND_RNG_PSEUDO_DEFAULT);
    // CHECK_CURAND(hiprandSetStream, rng_device_2, stream);
    // CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed, rng_device_2, rng_seed + proc_num * 2 + 1);

    // CHECK_CURAND(hiprandGenerateNormalDouble, rng_device_2, &((my_float_t*)(void*)state_data_device)[num_states_local], num_states_local, 0.0, 1.0);

    if (do_normalization) {

        if (proc_num == 0) { fprintf(stderr, "[info] gpu reduce\n"); } 
        {
            int64_t data_length = num_states_local;
            int64_t num_blocks_reduce;
            int block_size_reduce;

            if (data_length > block_size) {
                block_size_reduce = block_size;
                num_blocks_reduce = data_length >> log_block_size;
            } else {
                block_size_reduce = data_length;
                num_blocks_reduce = 1;
            }

            norm_sum_reduce_kernel<<<num_blocks_reduce, block_size_reduce, sizeof(my_float_t) * block_size_reduce, stream>>>(state_data_device, norm_sum_device);

            data_length = num_blocks_reduce;

            while (data_length > 1) {
                if (data_length > block_size) {
                    block_size_reduce = block_size;
                    num_blocks_reduce = data_length >> log_block_size;
                } else {
                    block_size_reduce = data_length;
                    num_blocks_reduce = 1;
                }

                sum_reduce_kernel<<<num_blocks_reduce, block_size_reduce, sizeof(my_float_t) * block_size_reduce, stream>>>(norm_sum_device, norm_sum_device);

                data_length = num_blocks_reduce;
            }
        }

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        my_float_t norm_sum_local;
        CHECK_CUDA(hipMemcpyAsync, &norm_sum_local, norm_sum_device, sizeof(my_float_t), hipMemcpyDeviceToHost, stream);

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        CHECK_CUDA(hipFree, (void*)norm_sum_device);

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        CHECK_CUDA(hipStreamSynchronize, stream);

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        my_float_t norm_sum_global;
        MPI_Allreduce(&norm_sum_local, &norm_sum_global, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
        if (proc_num == 0) { fprintf(stderr, "[info] norm_sum_global=%lf\n", norm_sum_global); }

        if (proc_num == 0) { fprintf(stderr, "[info] normalize\n"); }

        my_float_t const normalize_factor = 1.0 / sqrt(norm_sum_global);
        fprintf(stderr, "[debug] normalize_factor=%.20e\n", normalize_factor);

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        normalize_kernel<<<1ULL<<(num_qubits_local+1-log_block_size), block_size, 0, stream>>>((my_float_t*)(void*)state_data_device, normalize_factor);
        

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        CHECK_CUDA(hipEventRecord, event_2, stream);

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        CHECK_CUDA(hipStreamSynchronize, stream);

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        CHECK_CUDA(hipEventElapsedTime, &elapsed_ms, event_1, event_2);

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        MPI_Reduce(&elapsed_ms, &elapsed_ms_2, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
        elapsed_ms = elapsed_ms_2;

        // fprintf(stderr, "[debug] line=%d\n", __LINE__);

        if(proc_num==0) {
            fprintf(stderr, "[info] rng elapsed=%lf\n", elapsed_ms * 1e-3);
            fprintf(stderr, "[info] normalize done\n");
        }

    }

    if(proc_num==0) {
        fprintf(stderr, "[info] gpu_hadamard\n");
    }

    MPI_Barrier(MPI_COMM_WORLD);

    for(int sample_num=0; sample_num < num_samples; ++sample_num) {

        CHECK_CUDA(hipEventRecord, event_1, stream);

        for(int target_qubit_num_logical = target_qubit_num_begin; target_qubit_num_logical < target_qubit_num_end; target_qubit_num_logical++) {

            int target_qubit_num_physical = perm_l2p[target_qubit_num_logical];
            // if(proc_num==0) fprintf(stderr, "[debug] target_qubit_num_logical=%d target_qubit_num_physical=%d\n", target_qubit_num_logical, target_qubit_num_physical);
            // MPI_Barrier(MPI_COMM_WORLD);

            /* target qubits is global */
            if (target_qubit_num_physical >= num_qubits_local) {

                int const* const swap_target_global_list = &target_qubit_num_physical;
                int const swap_target_local = num_qubits - log_num_procs - 1;
                int const* const swap_target_local_list = &swap_target_local;
                int const num_targets = 1;

                // b_min
                int const swap_target_local_min = *std::min_element(swap_target_local_list, swap_target_local_list + num_targets);

                uint64_t const local_buf_length = UINT64_C(1) << swap_target_local_min;
                uint64_t swap_buffer_length = swap_buffer_total_length;
                if (swap_buffer_length > local_buf_length) {
                    swap_buffer_length = local_buf_length;
                }

                // generate a mask for generating global_nonswap_self
                uint64_t global_swap_self_mask = 0;
                for (int target_num = 0; target_num < num_targets; target_num++) {
                    // a_delta = a – n_local
                    int const swap_target_global_delta = swap_target_global_list[target_num] - num_qubits_local;
                    global_swap_self_mask |= (UINT64_C(1) << swap_target_global_delta);
                }

                // global_nonswap_self = make proc_num_self's a_delta_i-th digit zero
                uint64_t const global_nonswap_self = proc_num & ~global_swap_self_mask;

                // 1<<(num_local_qubits - b_min) 
                uint64_t const num_local_areas = UINT64_C(1) << (num_qubits_local - swap_target_local_min);
                for (uint64_t local_num_self = 0; local_num_self < num_local_areas; local_num_self++) {

                    // global_swap_peer = OR_i (local_num_selfのb_delta_i桁目)をa_delta_i桁目にする
                    uint64_t global_swap_peer = 0;
                    for (int target_num = 0; target_num < num_targets; target_num++) {
                        // a_delta_i
                        int const swap_target_global_delta = swap_target_global_list[target_num] - num_qubits_local;
                        // b_delta_i
                        int const swap_target_local_delta = swap_target_local_list[target_num] - swap_target_local_min;
                        global_swap_peer |=
                            // local_num_selfのb_delta_i桁目
                            ((local_num_self >> swap_target_local_delta) & 1)
                            // をa_delta_i桁目にする
                            << swap_target_global_delta;
                        
                    }

                    uint64_t const proc_num_peer = global_swap_peer | global_nonswap_self;

                    // send & recv
                    if (proc_num_peer == proc_num) { continue; }
                    // CHECK_NCCL(ncclSend, &state_data_device[local_num_self * local_buf_length], local_buf_length, ncclDouble, proc_num_peer, nccl_comm, stream);
                    // CHECK_NCCL(ncclRecv, &state_data_device[local_num_self * local_buf_length], local_buf_length, ncclDouble, proc_num_peer, nccl_comm, stream);
                    bool is_peer_greater = proc_num_peer > proc_num;
                    for (uint64_t buffer_pos = 0; buffer_pos < local_buf_length; buffer_pos += swap_buffer_length) {
                        CHECK_NCCL(ncclGroupStart);
                        for (int send_recv = 0; send_recv < 2; send_recv++) {
                            if (send_recv ^ is_peer_greater) {
                                CHECK_NCCL(ncclSend, &state_data_device[local_num_self * local_buf_length + buffer_pos], swap_buffer_length * 2 /* complex */, ncclDouble, proc_num_peer, nccl_comm, stream);
                            } else {
                                CHECK_NCCL(ncclRecv, swap_buffer, swap_buffer_length * 2 /* complex */, ncclDouble, proc_num_peer, nccl_comm, stream);
                            }
                        }
                        CHECK_NCCL(ncclGroupEnd);
                        CHECK_CUDA(hipMemcpyAsync, &state_data_device[local_num_self * local_buf_length + buffer_pos], swap_buffer, swap_buffer_length * sizeof(my_complex_t), hipMemcpyDeviceToDevice, stream);
                    }

                }

                // swap_target_global_logical_list[:] = perm_p2l[swap_target_global_list[:]]
                // swap_target_local_logical_list[:] = perm_p2l[swap_target_local_list[:]]
                std::vector<int> swap_target_local_logical_list(num_targets);
                std::vector<int> swap_target_global_logical_list(num_targets);
                for (int target_num = 0; target_num < num_targets; target_num++) {
                    swap_target_local_logical_list[target_num] = perm_p2l[swap_target_local_list[target_num]];
                    swap_target_global_logical_list[target_num] = perm_p2l[swap_target_global_list[target_num]];
                }

                // update p2l & l2p
                // perm_p2l[swap_target_global_list[:]] = swap_target_local_logical_list[:]
                // perm_p2l[swap_target_local_list[:]] = swap_target_global_logical_list[:]
                // perm_l2p[swap_target_global_logical_list[:]] = swap_target_local_list[:]
                // perm_l2p[swap_target_local_logical_list[:]] = swap_target_global_list[:]

                for (int target_num = 0; target_num < num_targets; target_num++) {
                    perm_p2l[swap_target_global_list[target_num]] = swap_target_local_logical_list[target_num];
                    perm_p2l[swap_target_local_list[target_num]] = swap_target_global_logical_list[target_num];
                    perm_l2p[swap_target_global_logical_list[target_num]] = swap_target_local_list[target_num];
                    perm_l2p[swap_target_local_logical_list[target_num]] = swap_target_global_list[target_num];
                }

                target_qubit_num_physical = swap_target_local;

            }

            cuda_gate<hadamard><<<num_blocks, block_size, 0, stream>>>(num_qubits, target_qubit_num_physical, state_data_device);
        }

        CHECK_CUDA(hipEventRecord, event_2, stream);

        CHECK_CUDA(hipStreamSynchronize, stream);

        CHECK_CUDA(hipEventElapsedTime, &elapsed_ms, event_1, event_2);
        MPI_Reduce(&elapsed_ms, &elapsed_ms_2, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);
        elapsed_ms = elapsed_ms_2;
        if (proc_num == 0) {
            fprintf(stderr, "[info] elapsed_gpu=%f\n", elapsed_ms * 1e-3);
            fprintf(stdout, "%lf\n", elapsed_ms * 1e-3);
        }

    }

    if (calc_checksum) {
        if (proc_num==0) {
            fprintf(stderr, "[info] gathering state data\n");

            EVP_MD_CTX *mdctx = EVP_MD_CTX_new();
            if (!mdctx) {
                perror("EVP_MD_CTX_new failed");
                exit(1);
            }
        
            if (EVP_DigestInit_ex(mdctx, EVP_sha256(), NULL) != 1) {
                perror("EVP_DigestInit_ex failed");
                EVP_MD_CTX_free(mdctx);
                exit(1);
            }

            my_complex_t* state_data_host = (my_complex_t*)malloc(num_states * sizeof(my_complex_t));
            DEFER_FUNC(free, state_data_host);

            CHECK_CUDA(hipMemcpyAsync, state_data_host, state_data_device, num_states_local * sizeof(my_complex_t), hipMemcpyDeviceToHost, stream);
            for(int peer_proc_num=1; peer_proc_num<num_procs; peer_proc_num++) {
                MPI_Status mpi_status;
                MPI_Recv(&state_data_host[peer_proc_num * num_states_local], num_states_local * 2, MPI_DOUBLE, peer_proc_num, 0, MPI_COMM_WORLD, &mpi_status);
            }
            CHECK_CUDA(hipStreamSynchronize, stream);

            for(int64_t state_num_logical = 0; state_num_logical < num_states; state_num_logical++) {
                int64_t state_num_physical = 0;
                for(int qubit_num_logical = 0; qubit_num_logical < num_qubits; qubit_num_logical++) {
                    int qubit_num_physical = perm_l2p[qubit_num_logical];
                    state_num_physical = state_num_physical | (((state_num_logical >> qubit_num_logical) & 1) << qubit_num_physical);
                }
                if (EVP_DigestUpdate(mdctx, &state_data_host[state_num_physical], sizeof(my_complex_t)) != 1) {
                    perror("EVP_DigestUpdate failed");
                    EVP_MD_CTX_free(mdctx);
                    exit(1);
                }
            }

            unsigned char evp_hash[EVP_MAX_MD_SIZE];
            unsigned int evp_hash_len;
            if (EVP_DigestFinal_ex(mdctx, evp_hash, &evp_hash_len) != 1) {
                perror("EVP_DigestFinal_ex failed");
                EVP_MD_CTX_free(mdctx);
                exit(1);
            }

            fprintf(stderr, "[info] checksum: ");
            for (unsigned int i = 0; i < evp_hash_len; i++) {
                fprintf(stderr, "%02x", evp_hash[i]);
            }
            fprintf(stderr, "\n");

            EVP_MD_CTX_free(mdctx);
        } else {
            MPI_Send(state_data_device, num_states_local * 2, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
        }
    }

    MPI_Finalize();

    return 0;

}